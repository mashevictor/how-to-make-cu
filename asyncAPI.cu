#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

//__device__ int addem( int a, int b ) {
  //  return a + b;
//}

__global__ void add( int a, int b, int *c ) {
    *c = a+b;
    //*c = addem( a, b );
}

int main( void ) {
    int c;
    int *dev_c;
    checkCudaErrors( hipMalloc( (void**)&dev_c, sizeof(int) ) );

    add<<<1,1>>>( 2000000, 7000000000, dev_c );

    checkCudaErrors( hipMemcpy( &c, dev_c, sizeof(int),
                              hipMemcpyDeviceToHost ) );
    printf( "2 + 7 = %d\n", c );
    checkCudaErrors( hipFree( dev_c ) );

    return 0;
}


